#include "hip/hip_runtime.h"
__global__ void add(int *a, int *b , int *result)
{
    int index=blockIdx.x * blockDim.x + threadIdx.x;
    *result[index]= *a[index] + *b[index];   
}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main()
{
    int a,b,result;
    int *device_a,*device_b,*device_result;
    int size=sizeof(int)*N;
    
    hipMalloc((void **)&device_a,size);
    hipMalloc((void **)&device_b,size);
    hipMalloc((void **)&device_result,size);
    
    a=(int *)malloc(size);
    b=(int *)malloc(size);
    result=(int *)malloc(size);
    
    random_ints(a,N);
    random_ints(b,N);
    
    hipMemcpy(device_a,&a,size,hipMemcpyHostToDevice);
    hipMemcpy(device_b,&b,size,hipMemcpyHostToDevice);
    
    add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(device_a,device_b,device_result);
    
    hipMemcpy(&result,device_result,size,hipMemcpyDeviceToHost);
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_result);
    
    free(a);
    free(b);
    free(result);
    
    return 0;
}




















