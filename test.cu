
#include <hip/hip_runtime.h>
__global__ void add(int *a, int *b , int *result)
{
    *result= *a + *b;   
}

int main()
{
    int a,b,result;
    int *device_a,*device_b,*device_result;
    int size=sizeof(int);
    
    hipMalloc((void **)&device_a,size);
    hipMalloc((void **)&device_b,size);
    hipMalloc((void **)&device_result,size);
    
    a=2;
    b=7;
    
    hipMemcpy(device_a,&a,size,hipMemcpyHostToDevice);
    hipMemcpy(device_b,&b,size,hipMemcpyHostToDevice);
    
    add<<<1,1>>>(device_a,device_b,device_result);
    
    hipMemcpy(&result,device_result,size,hipMemcpyDeviceToHost);
    
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_result);
    return 0;
}